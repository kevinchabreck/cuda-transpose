#include <stdio.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netdb.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <unistd.h>
#include <ctype.h>
#include <strings.h>
#include <string.h>
#include <sys/stat.h>
#include <pthread.h>
#include <sys/wait.h>
#include <stdlib.h>
#include <stdbool.h>
#include <assert.h>
#include <fcntl.h>
#include <sys/epoll.h>
#include <errno.h>
#include <hip/hip_runtime_api.h>
#include <nvToolsExt.h>
#include <nvToolsExtCuda.h>
#include <nvToolsExtCudaRt.h>
#include <time.h>
#include "common.h"
#include "common.c"
#include "memory.c"

hipStream_t stream;
char* h_idata;
char* h_odata;
char* d_idata;
char* d_odata;
dim3 grid;
dim3 threads;

void print_matrix(char * data, int x, int y){
  for( unsigned int i = 0; i < x; i++) {
    for( unsigned int j = 0; j < y; j++) {
      printf("%c", data[i*x + j]);
    }
    printf("\n");
  }
  printf("\n");
}

double throughput_bench(int request_size, int cohort_size, int iterations, int n){
  grid = dim3(request_size/TILE_DIM, cohort_size/TILE_DIM, 1);
  unsigned int mem_size = sizeof(char) * request_size * cohort_size;
  // allocate host memory
  h_idata = (char*) malloc(mem_size);
  h_odata = (char*) malloc(mem_size);
  // allocate device memory
  check_cuda( hipMalloc( (void**) &d_idata, mem_size));
  check_cuda( hipMalloc( (void**) &d_odata, mem_size));
  // initialize input matrix
  for( unsigned int i = 0; i < (request_size * cohort_size); i++) {
    h_idata[i] = (char) (i%26 + 97);
  }
  // initialize timer
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float times[iterations];
  for(int i=0; i<iterations; i++){
    // Tstart
    hipEventRecord(start, 0);
    for(int j=0; j<n; j++){
      // copy to device, transpose, and copy back to host
      check_cuda(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));
      transpose<<< grid, threads, 0, stream>>>(d_odata, d_idata, request_size, cohort_size);
      hipDeviceSynchronize();
      check_cuda(hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost));
    }
    // Tstop
    hipEventRecord(stop, 0);
    // calculate elapsed time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&times[i], start, stop);
  }
  double avg = 0;
  for(int i=0; i<iterations; i++){
    avg += times[i];
  }
  avg = avg / iterations;
  // free host mem
  free(h_idata);
  free(h_odata);
  // free device memory
  check_cuda( hipFree(d_idata));
  check_cuda( hipFree(d_odata));
  return avg;
}

double latency_bench(int request_size, int cohort_size, int iterations){
  grid = dim3(request_size/TILE_DIM, cohort_size/TILE_DIM, 1);

  unsigned int mem_size = sizeof(char) * request_size * cohort_size;
  // allocate host memory
  h_idata = (char*) malloc(mem_size);
  h_odata = (char*) malloc(mem_size);
  // allocate device memory
  check_cuda( hipMalloc( (void**) &d_idata, mem_size));
  check_cuda( hipMalloc( (void**) &d_odata, mem_size));
  // initialize input matrix
  for( unsigned int i = 0; i < (request_size * cohort_size); i++) {
    h_idata[i] = (char) (i%26 + 97);
  }
  // initialize timer
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float times[iterations];
  for(int i=0; i<iterations; i++){
    // Tstart
    hipEventRecord(start, 0);
    // copy to device, transpose, and copy back to host
    check_cuda(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));
    transpose<<< grid, threads, 0, stream>>>(d_odata, d_idata, request_size, cohort_size);
    hipDeviceSynchronize();
    check_cuda(hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost));
    // Tstop
    hipEventRecord(stop, 0);
    // calculate elapsed time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&times[i], start, stop);
  }
  double avg = 0;
  for(int i=0; i<iterations; i++){
    avg += times[i];
  }
  avg = avg / iterations;
  // free host mem
  free(h_idata);
  free(h_odata);
  // free device memory
  check_cuda( hipFree(d_idata));
  check_cuda( hipFree(d_odata));
  return avg;
}

void run_benchmarks(int request_size, int cohort_size, int iterations){
  // set up execution environment
  threads = dim3(TILE_DIM, BLOCK_ROWS, 1);
  stream = alloc_stream();
  // run benchmarks
  int n = 100; // number of transposes to test throughput with
  double throughput = throughput_bench(request_size, cohort_size, iterations, n);
  printf("\n*THROUGHPUT BENCHMARK*\n");
  printf("iterations: %d\n", iterations);
  printf("transposes per iteration: %d\n", n);
  printf("request size: %d\n", request_size);
  printf("cohort size: %d\n", cohort_size);
  printf("average time for %d transposes (ms): %f\n", n, throughput);
  double latency = latency_bench(request_size, cohort_size, iterations);
  printf("\n*LATENCY BENCHMARK*\n");
  printf("iterations: %d\n", iterations);
  printf("request size: %d\n", request_size);
  printf("cohort size: %d\n", cohort_size);
  printf("average latency (ms): %f\n", latency);
}

int main(void)
{ 
  int request_size = 1024; // length of requests in bytes
  int cohort_size  = 4096; // # of requests in cohorts (shouldnt change)
  int iterations   = 100;  // number of iterations to run benchmarks
  run_benchmarks(request_size, cohort_size, iterations);
  
  hipDeviceReset();
}
